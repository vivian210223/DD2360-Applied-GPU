#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// includes, kernels
#include "backprop_cuda_kernel.cu"
#include "backprop.h"

////////////////////////////////////////////////////////////////////////////////

extern "C"
void bpnn_layerforward(float *l1, float *l2, float **conn, int n1, int n2);

extern "C"
void bpnn_output_error(float *delta, float *target, float *output, int nj, float *err);

extern "C"
void bpnn_hidden_error(float *delta_h, int nh, float *delta_o, int no, float **who, float *hidden, float *err);

extern "C" 
void bpnn_adjust_weights(float *delta, int ndelta, float *ly, int nly, float **w, float **oldw);


extern "C"
int setup(int argc, char** argv);

extern "C"
float **alloc_2d_dbl(int m, int n);

extern "C"
float squash(float x);

/*double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}*/

unsigned int num_threads = 0;
unsigned int num_blocks = 0;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
  setup(argc, argv);
}


extern "C"
void bpnn_train_cuda(BPNN *net, float *eo, float *eh)
{
  int in, hid, out;
  float out_err, hid_err;
  
  in = net->input_n;
  hid = net->hidden_n;
  out = net->output_n;   

#ifdef GPU  
  int m = 0;
  float *input_hidden_cuda;
  float *input_cuda;
  float *output_hidden_cuda;
  float *partial_sum;
  float *hidden_partial_sum;
  float *hidden_delta_cuda;
  float *input_prev_weights_cuda;
  float sum;
  float *input_weights_one_dim;
  float *input_weights_prev_one_dim;
  num_blocks = in / 16;  
  dim3  grid( 1 , num_blocks);
  dim3  threads(16 , 16);
  
  input_weights_one_dim = (float *) malloc((in + 1)* (hid + 1) * sizeof(float));
  input_weights_prev_one_dim = (float *) malloc((in + 1)* (hid + 1) * sizeof(float));
  partial_sum = (float *) malloc(num_blocks * WIDTH * sizeof(float));
 
  // this preprocessing stage is added to correct the bugs of wrong memcopy using two-dimensional net->inputweights
  for (int k = 0; k <= in; k++) {	
   for (int j = 0; j <= hid; j++) {
	  input_weights_one_dim[m] = net->input_weights[k][j];
	  input_weights_prev_one_dim[m] = net-> input_prev_weights[k][j];
	  m++;
    }
  }
  hipMalloc((void**) &input_cuda, (in + 1) * sizeof(float));
  hipMalloc((void**) &output_hidden_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float));
  hipMalloc((void**) &hidden_partial_sum, num_blocks * WIDTH * sizeof(float));

#endif

#ifdef UM
   
  int m = 0;
  float *partial_sum;
  //float *input_cuda;
  float *output_hidden_cuda;
  float sum;
  float *input_weights_one_dim;
  float *input_weights_prev_one_dim;
  num_blocks = in / 16;  
  dim3  grid( 1 , num_blocks);
  dim3  threads(16 , 16);
  
  
  hipMallocManaged((void**)&input_weights_one_dim, (in + 1)* (hid + 1) * sizeof(float));
  hipMallocManaged((void**)&input_weights_prev_one_dim, (in + 1)* (hid + 1) * sizeof(float));
  hipMallocManaged((void**)&partial_sum, num_blocks * WIDTH * sizeof(float));
  hipMallocManaged((void**)&output_hidden_cuda, (hid + 1) * sizeof(float));
  //hipMalloc((void**) &input_cuda, (in + 1) * sizeof(float));

  // this preprocessing stage is added to correct the bugs of wrong memcopy using two-dimensional net->inputweights
  for (int k = 0; k <= in; k++) {	
   for (int j = 0; j <= hid; j++) {
	  input_weights_one_dim[m] = net->input_weights[k][j];
	  input_weights_prev_one_dim[m] = net-> input_prev_weights[k][j];
	  m++;
    }
  }

#endif

#ifdef PM  
  int m = 0;
  float *input_hidden_cuda;
  float *input_cuda;
  float *output_hidden_cuda;
  float *partial_sum;
  float *hidden_partial_sum;
  float *hidden_delta_cuda;
  float *input_prev_weights_cuda;
  float sum;
  float *input_weights_one_dim;
  float *input_weights_prev_one_dim;
  num_blocks = in / 16;  
  dim3  grid( 1 , num_blocks);
  dim3  threads(16 , 16);
  
  hipHostMalloc((void**)&input_weights_one_dim, (in + 1)* (hid + 1) * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)&input_weights_prev_one_dim, (in + 1)* (hid + 1) * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)&partial_sum, num_blocks * WIDTH * sizeof(float), hipHostMallocDefault);
  
  // this preprocessing stage is added to correct the bugs of wrong memcopy using two-dimensional net->inputweights
  for (int k = 0; k <= in; k++) {	
   for (int j = 0; j <= hid; j++) {
	  input_weights_one_dim[m] = net->input_weights[k][j];
	  input_weights_prev_one_dim[m] = net-> input_prev_weights[k][j];
	  m++;
    }
  }
  hipMalloc((void**) &input_cuda, (in + 1) * sizeof(float));
  hipMalloc((void**) &output_hidden_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float));
  hipMalloc((void**) &hidden_partial_sum, num_blocks * WIDTH * sizeof(float));

#endif

#ifdef ZC
   
  int m = 0;
  float *input_hidden_cuda;
  float *input_cuda;
  float *output_hidden_cuda;
  float *partial_sum;
  float *hidden_partial_sum;
  float *hidden_delta_cuda;
  float *input_prev_weights_cuda;
  float sum;
  float *input_weights_one_dim;
  float *input_weights_prev_one_dim;
  num_blocks = in / 16;  
  dim3  grid( 1 , num_blocks);
  dim3  threads(16 , 16);
  
  hipHostAlloc((void**)&input_weights_one_dim, (in + 1)* (hid + 1) * sizeof(float), hipHostMallocMapped);
  hipHostAlloc((void**)&input_weights_prev_one_dim, (in + 1)* (hid + 1) * sizeof(float), hipHostMallocMapped);
  hipHostAlloc((void**)&partial_sum, num_blocks * WIDTH * sizeof(float), hipHostMallocMapped);
  hipMalloc((void**) &output_hidden_cuda, (hid + 1) * sizeof(float));
  
  // this preprocessing stage is added to correct the bugs of wrong memcopy using two-dimensional net->inputweights
  for (int k = 0; k <= in; k++) {	
   for (int j = 0; j <= hid; j++) {
	  input_weights_one_dim[m] = net->input_weights[k][j];
	  input_weights_prev_one_dim[m] = net-> input_prev_weights[k][j];
	  m++;
    }
  }
  
#endif

#ifdef CPU

  printf("Performing CPU computation\n");
  bpnn_layerforward(net->input_units, net->hidden_units,net->input_weights, in, hid);

#endif

#ifdef GPU
 
  printf("Performing GPU computation\n");

  hipMemcpy(input_cuda, net->input_units, (in + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  
  bpnn_layerforward_CUDA<<< grid, threads >>>(input_cuda,
	                                          output_hidden_cuda,
											  input_hidden_cuda,
											  hidden_partial_sum,
											  in,
											  hid);
 
  hipDeviceSynchronize();
  
  hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("bpnn kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
  
  hipMemcpy(partial_sum, hidden_partial_sum, num_blocks * WIDTH * sizeof(float), hipMemcpyDeviceToHost);
     
  for (int j = 1; j <= hid; j++) {
    sum = 0.0;
    for (int k = 0; k < num_blocks; k++) {	
      sum += partial_sum[k * hid + j-1] ;
    }
	sum += net->input_weights[0][j];
	net-> hidden_units[j] = float(1.0 / (1.0 + exp(-sum)));
  }
#endif

#ifdef UM
  
  printf("Performing unified memory GPU computation\n");
  
  //hipMemcpy(input_cuda, net->input_units, (in + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemPrefetchAsync(net->input_units, (in + 1) * sizeof(float), 0);
  hipMemPrefetchAsync(output_hidden_cuda, (hid+1) * sizeof(float), 0);
  hipMemPrefetchAsync(input_weights_one_dim, (in+1) * (hid+1) * sizeof(float), 0);
  hipMemPrefetchAsync(partial_sum, num_blocks * WIDTH * sizeof(float), 0);

  bpnn_layerforward_CUDA<<< grid, threads >>>(net->input_units,
	                                          output_hidden_cuda,
											  input_weights_one_dim,
											  partial_sum,
											  in,
											  hid);
  
  hipDeviceSynchronize();
  

  hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("bpnn kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
  //hipMemPrefetchAsync(net->input_weights, (in + 1) * hid * sizeof(float), 0);

      
  for (int j = 1; j <= hid; j++) {
    sum = 0.0;
    for (int k = 0; k < num_blocks; k++) {	
      sum += partial_sum[k * hid + j-1] ;
    }
	sum += net->input_weights[0][j];
	net-> hidden_units[j] = float(1.0 / (1.0 + exp(-sum)));
  }

#endif

#ifdef PM
 
  printf("Performing pin memory computation\n");

  hipMemcpy(input_cuda, net->input_units, (in + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  
  bpnn_layerforward_CUDA<<< grid, threads >>>(input_cuda,
	                                          output_hidden_cuda,
											  input_hidden_cuda,
											  hidden_partial_sum,
											  in,
											  hid);
 
  hipDeviceSynchronize();
  
  hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("bpnn kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
  
  hipMemcpy(partial_sum, hidden_partial_sum, num_blocks * WIDTH * sizeof(float), hipMemcpyDeviceToHost);
     
  for (int j = 1; j <= hid; j++) {
    sum = 0.0;
    for (int k = 0; k < num_blocks; k++) {	
      sum += partial_sum[k * hid + j-1] ;
    }
	sum += net->input_weights[0][j];
	net-> hidden_units[j] = float(1.0 / (1.0 + exp(-sum)));
  }
#endif

#ifdef ZC
 
  printf("Performing zero copy computation\n");
  hipHostGetDevicePointer(&input_cuda, net->input_units, 0);
  hipHostGetDevicePointer(&input_hidden_cuda, input_weights_one_dim, 0);
  hipHostGetDevicePointer(&hidden_partial_sum, partial_sum, 0);
  
  bpnn_layerforward_CUDA<<< grid, threads >>>(input_cuda,
	                                          output_hidden_cuda,
											  input_hidden_cuda,
											  hidden_partial_sum,
											  in,
											  hid);
 
  hipDeviceSynchronize();
  
  hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("bpnn kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
     
  for (int j = 1; j <= hid; j++) {
    sum = 0.0;
    for (int k = 0; k < num_blocks; k++) {	
      sum += partial_sum[k * hid + j-1] ;
    }
	sum += net->input_weights[0][j];
	net-> hidden_units[j] = float(1.0 / (1.0 + exp(-sum)));
  }

  
#endif

  bpnn_layerforward(net->hidden_units, net->output_units, net->hidden_weights, hid, out);
  bpnn_output_error(net->output_delta, net->target, net->output_units, out, &out_err);
  bpnn_hidden_error(net->hidden_delta, hid, net->output_delta, out, net->hidden_weights, net->hidden_units, &hid_err);  
  bpnn_adjust_weights(net->output_delta, out, net->hidden_units, hid, net->hidden_weights, net->hidden_prev_weights);
  *eo = out_err;
  *eh = hid_err;

#ifdef CPU

  bpnn_adjust_weights(net->hidden_delta, hid, net->input_units, in, net->input_weights, net->input_prev_weights);

#endif  

#ifdef GPU

  hipMalloc((void**) &hidden_delta_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_prev_weights_cuda, (in + 1) * (hid + 1) * sizeof(float));

  hipMemcpy(hidden_delta_cuda, net->hidden_delta, (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_prev_weights_cuda, input_weights_prev_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);


  bpnn_adjust_weights_cuda<<< grid, threads >>>(hidden_delta_cuda,  
												hid, 
												input_cuda, 
												in,
												input_hidden_cuda, 
												input_prev_weights_cuda
												);

  hipMemcpy(net->input_units, input_cuda, (in + 1) * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(input_weights_one_dim, input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyDeviceToHost);
    
  hipFree(input_cuda);
  hipFree(output_hidden_cuda);
  hipFree(input_hidden_cuda);
  hipFree(hidden_partial_sum);
  hipFree(input_prev_weights_cuda);
  hipFree(hidden_delta_cuda);
  
  free(partial_sum);
  free(input_weights_one_dim);
  free(input_weights_prev_one_dim);

#endif   

#ifdef UM

  //float *hidden_delta_cuda;
  //hipMalloc((void**) &hidden_delta_cuda, (hid + 1) * sizeof(float));
  //hipMemcpy(hidden_delta_cuda, net->hidden_delta, (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemPrefetchAsync(net->hidden_delta, (hid+1) * sizeof(float), 0);
  hipMemPrefetchAsync(net->input_units, (in + 1) * sizeof(float), 0);
  hipMemPrefetchAsync(input_weights_one_dim, (in+1) * (hid+1) * sizeof(float), 0);
  hipMemPrefetchAsync(input_weights_prev_one_dim, (in+1) * (hid+1) * sizeof(float), 0);

  bpnn_adjust_weights_cuda<<< grid, threads >>>(net->hidden_delta,  
												hid, 
												net->input_units, 
												in,
												input_weights_one_dim, 
												input_weights_prev_one_dim
												);

  
  //hipFree(input_cuda);
  //hipFree(hidden_delta_cuda);  
  hipFree(output_hidden_cuda);
  hipFree(partial_sum);
  hipFree(input_weights_one_dim);
  hipFree(input_weights_prev_one_dim);

#endif  

#ifdef PM

  hipMalloc((void**) &hidden_delta_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_prev_weights_cuda, (in + 1) * (hid + 1) * sizeof(float));

  hipMemcpy(hidden_delta_cuda, net->hidden_delta, (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_prev_weights_cuda, input_weights_prev_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);


  bpnn_adjust_weights_cuda<<< grid, threads >>>(hidden_delta_cuda,  
												hid, 
												input_cuda, 
												in,
												input_hidden_cuda, 
												input_prev_weights_cuda
												);

  hipMemcpy(net->input_units, input_cuda, (in + 1) * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(input_weights_one_dim, input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyDeviceToHost);
    
  hipFree(input_cuda);
  hipFree(output_hidden_cuda);
  hipFree(input_hidden_cuda);
  hipFree(hidden_partial_sum);
  hipFree(input_prev_weights_cuda);
  hipFree(hidden_delta_cuda);
  
  hipHostFree(partial_sum);
  hipHostFree(input_weights_one_dim);
  hipHostFree(input_weights_prev_one_dim);

#endif   

#ifdef ZC

  hipHostGetDevicePointer((void **)&hidden_delta_cuda, net->hidden_delta, 0);
  hipHostGetDevicePointer((void **)&input_prev_weights_cuda, input_weights_prev_one_dim, 0);
  hipHostGetDevicePointer((void **)&input_hidden_cuda, input_weights_one_dim, 0);

  

  bpnn_adjust_weights_cuda<<< grid, threads >>>(hidden_delta_cuda,  
												hid, 
												input_cuda, 
												in,
												input_hidden_cuda, 
												input_prev_weights_cuda
												);

  

  
  hipFree(hidden_delta_cuda);  
  hipFree(output_hidden_cuda);
  hipFree(input_hidden_cuda);
  hipFree(partial_sum);
  hipFree(input_weights_one_dim);
  hipFree(input_weights_prev_one_dim);

#endif  

}
