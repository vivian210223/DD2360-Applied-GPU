#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>

#define DataType double
//#define nStreams 8

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  
  // acquire index first
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // vector addition
  if (id < len){
  	out[id] = in1[id]+in2[id];
  }
  
}

//@@ Insert code to implement timer start
void timeStart(struct timeval *startt)
{
  gettimeofday(startt,NULL);
}
//@@ Insert code to implement timer stop
void timeStop(struct timeval *startt)
{
  struct timeval stopt;
  float timet;
  gettimeofday(&stopt,NULL);
  timet = (stopt.tv_usec-startt->tv_usec)*1.0e-6 + stopt.tv_sec - startt->tv_sec;
  printf("Time: %f (s)\n",timet);
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;


  //@@ Insert code below to read in inputLength from args
  inputLength = atoi(argv[1]);
  int nStreams = atoi(argv[3]);
  const int streamSize = inputLength/nStreams;
  const int streamBytes = streamSize * sizeof(DataType);
  
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  //hipHostAlloc(&hostInput1, inputLength*sizeof(DataType), hipHostMallocDefault);
  //hipHostAlloc(&hostInput2, inputLength*sizeof(DataType), hipHostMallocDefault);
  hostInput1 = (DataType *)malloc(inputLength*sizeof(DataType));
  hostInput2 = (DataType *)malloc(inputLength*sizeof(DataType));
  hostOutput = (DataType *) malloc(inputLength*sizeof(DataType));
  resultRef = (DataType *) malloc(inputLength*sizeof(DataType));
  hipStream_t stream[nStreams];
  for (int i=0; i<nStreams; ++i){
  	hipStreamCreate(&stream[i]);
  }

  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers
  //and create reference result in CPU
  
  // Use current time as 
  // seed for random generator 
  srand(time(0));  
  for (int i=0; i<inputLength; i++){
  	hostInput1[i] = rand();
  	hostInput2[i] = rand();
  	resultRef[i] = hostInput1[i]+hostInput2[i];		
  }

  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength*sizeof(DataType));
  
  struct timeval timekernel;
  timeStart(&timekernel);
  for (int i=0; i<nStreams; ++i){
  	int offset = i*streamSize;
    //@@ Insert code to below to Copy memory to the GPU here
  	hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], 
  				streamBytes, hipMemcpyHostToDevice, stream[i]);
  	hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset], 
  				streamBytes, hipMemcpyHostToDevice, stream[i]);
    
    //@@ Initialize the 1D grid and block dimensions here
    int TPB = atoi(argv[2]);
    int BPG = (streamSize + TPB - 1)/TPB; 

    //@@ Launch the GPU Kernel here
    // mykernel<<<blocks, threads, shared_mem, stream>>>(args);
    vecAdd<<<BPG, TPB, 0, stream[i]>>>
    (&deviceInput1[offset], &deviceInput2[offset], &deviceOutput[offset], streamSize);

    //@@ Copy the GPU memory back to the CPU here
    hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset],
  					streamBytes, hipMemcpyDeviceToHost, stream[i]);
  }
  hipDeviceSynchronize();
  timeStop(&timekernel);

  //@@ Insert code below to compare the output with the reference
  int equal = 1;
  for(int i=0; i<inputLength; i++){
  	if (hostOutput[i] == resultRef[i]){
  		continue;
  	}
    else{
      printf("The result is not equal in %dth vector element.\n",i);
      printf("hostOutput: %lf and reference: %lf", hostOutput[i], resultRef[i]);
      equal = 0;
    }
  }
  if (equal){
    printf("The output is equal to the reference");
  } 
  	

  //@@ Free the GPU memory here
  for (int i=0; i<nStreams; i++){
  	hipStreamDestroy(stream[i]);
  }
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);
  
  //@@ Free the CPU memory here
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  free(resultRef);
  
  return 0;
}
